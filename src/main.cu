#include "hip/hip_runtime.h"
#include "color.h"
#include "fps.h"
#include "SM.h"
#include "framebufferConfig.h"
#include <glm/glm.hpp>

Color Background = {0, 0, 0};
const float ASPECT_RATIO = static_cast<float>(SCREEN_WIDTH) / static_cast<float>(SCREEN_HEIGHT);

__global__ void render(Point *buffer)
{
  
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int x = index % SCREEN_WIDTH;
  int y = index / SCREEN_WIDTH;

  float screenX =  ((2.0f * x) / SCREEN_WIDTH) - 1.0f;
  float screenY = -((2.0f * y) / SCREEN_HEIGHT) + 1.0f;

  if(screenX > 1.0f || screenX < -1.0f){
    printf("Me he pasado en X \n");
    return;
  }

  if(screenY > 1.0f || screenY < -1.0f){
    printf("Me he pasado en Y \n");
    return;
  }

  Point p = {x, y, 0, Color((screenX + 1.0f) / 2.0f, 0.0f, (screenY + 1.0f) / 2.0f)};

  buffer[index] = p;

}

int main(int argc, char *argv[])
{

  /*-------------------------CUDA CONFIGURATION----------------------------*/

  int deviceCount;
  int numCores;
  hipGetDeviceCount(&deviceCount);

  if (deviceCount > 0)
  {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    numCores = std::min(deviceProp.multiProcessorCount * _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor), 1024);
  }

  int numBlocks = (SCREEN_WIDTH * SCREEN_HEIGHT + numCores - 1) / numCores;

  /*-------------------------SDL CONFIGURATION----------------------------*/

  SDL_Init(SDL_INIT_VIDEO);

  SDL_Window *window = SDL_CreateWindow(
      "SDL2Test",
      SDL_WINDOWPOS_UNDEFINED,
      SDL_WINDOWPOS_UNDEFINED,
      SCREEN_WIDTH,
      SCREEN_HEIGHT,
      0);

  SDL_Renderer *renderer = SDL_CreateRenderer(window, -1, SDL_RENDERER_SOFTWARE);
  /*----------------------------------------------------------------------*/

  bool running = true;

  while (running)
  {
    startFPS();

    SDL_Event event;

    while (SDL_PollEvent(&event))
    {
      if (event.type == SDL_QUIT)
      {
        running = false;
      }
    }
    SDL_SetRenderDrawColor(renderer, Background.getRed(), Background.getGreen(), Background.getBlue(), SDL_ALPHA_OPAQUE);
    SDL_RenderClear(renderer);

    initBuffer();

    render<<<numBlocks, numCores>>>(dev_buffer);
    hipDeviceSynchronize();
    hipMemcpy(host_buffer, dev_buffer, SCREEN_WIDTH * SCREEN_HEIGHT * sizeof(Point), hipMemcpyDeviceToHost);

    renderBuffer(renderer, host_buffer);

    destroyBuffer();

    endFPS(window);
  }

  SDL_DestroyWindow(window);
  SDL_Quit();

  return 0;
}
