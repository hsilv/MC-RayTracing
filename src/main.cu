#include "hip/hip_runtime.h"
#include "color.h"
#include "fps.h"
#include "SM.h"
#include "framebufferConfig.h"
#include <glm/glm.hpp>
#include "ray.h"
#include "object.h"
#include <vector>
#include <thrust/device_vector.h>

Color Background = {0, 0, 0};
const float ASPECT_RATIO = static_cast<float>(SCREEN_WIDTH) / static_cast<float>(SCREEN_HEIGHT);

/* std::vector<Object> objects; */
thrust::device_vector<ObjectWrapper> objects;
std::vector<Material *> matPointers;

__global__ void render(Point *buffer, ObjectWrapper *objects, int numObjects)
{
  float fov = FOV;
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int x = index % SCREEN_WIDTH;
  int y = index / SCREEN_WIDTH;

  float screenX = ((2.0f * (x + 0.5f)) / SCREEN_WIDTH) - 1.0f;
  float screenY = -((2.0f * (y + 0.5f)) / SCREEN_HEIGHT) + 1.0f;
  screenX *= ASPECT_RATIO;
  screenX *= tan(fov / 2.0f);
  screenY *= tan(fov / 2.0f);

  glm::vec3 rayDirection = glm::normalize(glm::vec3(screenX, screenY, -1.0f));
  Color pixelColor = castRay(glm::vec3(0.0f, 0.0f, 0.0f), rayDirection, objects, numObjects);
  Point p = {x, y, 0, pixelColor};

  buffer[index] = p;
}

void setUp()
{

  Material *dev_rubber;
  hipMalloc(&dev_rubber, sizeof(Material));
  Material tempRubber = Material{Color(80, 0, 0)};
  hipMemcpy(dev_rubber, &tempRubber, sizeof(Material), hipMemcpyHostToDevice);

  matPointers.push_back(dev_rubber);

  Material *dev_ivory;
  hipMalloc(&dev_ivory, sizeof(Material));
  Material tempIvory = Material{Color(100, 100, 80)};
  hipMemcpy(dev_ivory, &tempIvory, sizeof(Material), hipMemcpyHostToDevice);

  matPointers.push_back(dev_ivory);


  Sphere *dev_sphere;
  hipMalloc(&dev_sphere, sizeof(Sphere));
  Sphere tempSphere = Sphere(glm::vec3(0.0f, 0.0f, -5.0f), 1.0f, tempRubber);
  hipMemcpy(dev_sphere, &tempSphere, sizeof(Sphere), hipMemcpyHostToDevice);

  ObjectWrapper sphereWrapper1;

  sphereWrapper1.obj = dev_sphere;
  sphereWrapper1.type = ObjectType::SPHERE;


  Sphere *dev_sphere2;
  hipMalloc(&dev_sphere2, sizeof(Sphere));
  Sphere tempSphere2 = Sphere(glm::vec3(-1.0f, 0.0f, -3.5f), 1.0f, tempIvory);
  hipMemcpy(dev_sphere2, &tempSphere2, sizeof(Sphere), hipMemcpyHostToDevice);

  ObjectWrapper sphereWrapper2;

  sphereWrapper2.obj = dev_sphere2;
  sphereWrapper2.type = ObjectType::SPHERE;

  objects.push_back(sphereWrapper1);
  objects.push_back(sphereWrapper2);
}

void destroy()
{

  while (objects.size() == 0)
  {
    ObjectWrapper obj = objects.back();
    hipFree(obj.obj);
    objects.pop_back();
  }

  while(matPointers.size() == 0){
    Material *mat = matPointers.back();
    hipFree(mat);
    matPointers.pop_back();
  }
}

int main(int argc, char *argv[])
{

  /*-------------------------CUDA CONFIGURATION----------------------------*/

  int deviceCount;
  int numCores;
  hipGetDeviceCount(&deviceCount);

  if (deviceCount > 0)
  {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    numCores = std::min(deviceProp.multiProcessorCount * _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor), 1024);
  }

  int numBlocks = (SCREEN_WIDTH * SCREEN_HEIGHT + numCores - 1) / numCores;

  /*-------------------------SDL CONFIGURATION----------------------------*/

  SDL_Init(SDL_INIT_VIDEO);

  SDL_Window *window = SDL_CreateWindow(
      "SDL2Test",
      SDL_WINDOWPOS_UNDEFINED,
      SDL_WINDOWPOS_UNDEFINED,
      SCREEN_WIDTH,
      SCREEN_HEIGHT,
      0);

  SDL_Renderer *renderer = SDL_CreateRenderer(window, -1, SDL_RENDERER_SOFTWARE);
  /*----------------------------------------------------------------------*/

  bool running = true;

  setUp();

  while (running)
  {
    startFPS();

    SDL_Event event;

    while (SDL_PollEvent(&event))
    {
      if (event.type == SDL_QUIT)
      {
        running = false;
      }
    }
    SDL_SetRenderDrawColor(renderer, Background.getRed(), Background.getGreen(), Background.getBlue(), SDL_ALPHA_OPAQUE);
    SDL_RenderClear(renderer);

    initBuffer();

    ObjectWrapper *raw_ptr = thrust::raw_pointer_cast(objects.data());

    render<<<numBlocks, numCores>>>(dev_buffer, raw_ptr, objects.size());
    hipDeviceSynchronize();
    hipMemcpy(host_buffer, dev_buffer, SCREEN_WIDTH * SCREEN_HEIGHT * sizeof(Point), hipMemcpyDeviceToHost);

    renderBuffer(renderer, host_buffer);

    destroyBuffer();

    endFPS(window);
  }

  destroy();

  SDL_DestroyWindow(window);
  SDL_Quit();

  return 0;
}
