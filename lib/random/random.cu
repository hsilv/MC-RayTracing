#include "hip/hip_runtime.h"
#include "random.h"
#include <hip/hip_runtime.h>

__device__ hiprandState_t state;


__global__ void random_init(unsigned long long seed) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(seed, idx, 0, &state);
}

__device__ float randomC() {
    return hiprand_uniform(&state);
}